#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define MATRIX_SIZE_X 5
#define MATRIX_SIZE_Y 5
#define PADDING_SIZE_X 1
#define BLOCK_SIZE_X 16
#define BLOCK_SIZE_Y 16

// Kernel para aplicar el relleno (padding)
__global__ void applyPaddingKernel(int *inputMatrix, int *paddedMatrix, int width, int height, int paddingX) {
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < width && row < height) {
        paddedMatrix[row * (width + paddingX) + col + paddingX] = inputMatrix[row * width + col];
    }
}

// Kernel para sumar las columnas
__global__ void sumColumnsKernel(int *matrix, int *result, int width, int height) {
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (col < width) {
        int sum = 0;
        for (int row = 0; row < height; ++row) {
            sum += matrix[row * width + col];
        }
        result[col] = sum;
    }
}

// Función para imprimir una matriz
void printMatrix(int *matrix, int rows, int cols, const char *label) {
    printf("%s:\n", label);
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            printf("%d\t", matrix[i * cols + j]);
        }
        printf("\n");
    }
}

int main() {
    int originalMatrix[MATRIX_SIZE_X][MATRIX_SIZE_Y];
    int paddedMatrix[(MATRIX_SIZE_X + PADDING_SIZE_X) * MATRIX_SIZE_Y];
    int columnSumResult[MATRIX_SIZE_Y];

    // Inicialización de la matriz original
    for (int i = 0; i < MATRIX_SIZE_X; ++i) {
        for (int j = 0; j < MATRIX_SIZE_Y; ++j) {
            originalMatrix[i][j] = (rand() % 9) + 1;
        }
    }

    // Imprime la matriz original
    printMatrix((int *)originalMatrix, MATRIX_SIZE_X, MATRIX_SIZE_Y, "Original");

    // Variables para GPU
    int *d_originalMatrix, *d_paddedMatrix, *d_columnSumResult;

    // Asignación de memoria en GPU
    hipMalloc((void **)&d_originalMatrix, MATRIX_SIZE_X * MATRIX_SIZE_Y * sizeof(int));
    hipMalloc((void **)&d_paddedMatrix, (MATRIX_SIZE_X + PADDING_SIZE_X) * MATRIX_SIZE_Y * sizeof(int));
    hipMalloc((void **)&d_columnSumResult, MATRIX_SIZE_Y * sizeof(int));

    // Copia la matriz original a la GPU
    hipMemcpy(d_originalMatrix, originalMatrix, MATRIX_SIZE_X * MATRIX_SIZE_Y * sizeof(int), hipMemcpyHostToDevice);

    // Definición de las dimensiones de bloque y cuadrícula
    dim3 gridDim((MATRIX_SIZE_Y + BLOCK_SIZE_X - 1) / BLOCK_SIZE_X, (MATRIX_SIZE_X + BLOCK_SIZE_Y - 1) / BLOCK_SIZE_Y, 1);
    dim3 blockDim(BLOCK_SIZE_X, BLOCK_SIZE_Y, 1);

    // Llama al kernel para aplicar el relleno
    applyPaddingKernel<<<gridDim, blockDim>>>(d_originalMatrix, d_paddedMatrix, MATRIX_SIZE_Y, MATRIX_SIZE_X, PADDING_SIZE_X);

    // Copia los resultados de GPU a CPU
    hipMemcpy(paddedMatrix, d_paddedMatrix, (MATRIX_SIZE_X + PADDING_SIZE_X) * MATRIX_SIZE_Y * sizeof(int), hipMemcpyDeviceToHost);

    // Imprime la matriz con relleno
    printMatrix(paddedMatrix, MATRIX_SIZE_X + PADDING_SIZE_X, MATRIX_SIZE_Y, "Con Relleno");

    // Llama al kernel para sumar las columnas
    sumColumnsKernel<<<gridDim, blockDim>>>(d_paddedMatrix, d_columnSumResult, MATRIX_SIZE_Y, MATRIX_SIZE_X + PADDING_SIZE_X);

    // Copia los resultados de la suma de columnas de GPU a CPU
    hipMemcpy(columnSumResult, d_columnSumResult, MATRIX_SIZE_Y * sizeof(int), hipMemcpyDeviceToHost);

    // Imprime las sumas de las columnas
    printMatrix(columnSumResult, 1, MATRIX_SIZE_Y, "Suma de Columnas");

    // Libera la memoria de GPU
    hipFree(d_originalMatrix);
    hipFree(d_paddedMatrix);
    hipFree(d_columnSumResult);

    return 0;
}
